/*
//@HEADER
// *****************************************************************************
//
//  HPCGraph: Graph Computation on High Performance Computing Systems
//              Copyright (2016) Sandia Corporation
//
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions?  Contact  George M. Slota   (gmslota@sandia.gov)
//                      Siva Rajamanickam (srajama@sandia.gov)
//                      Kamesh Madduri    (madduri@cse.psu.edu)
//
// *****************************************************************************
//@HEADER
*/

#include <mpi.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "comms.h"
#include "util.h"

extern int procid, nprocs;
extern bool verbose, debug, verify, output;

void init_queue_data(dist_graph_t* g, queue_data_t* q)
{  
  if (debug) { printf("Task %d init_queue_data() start\n", procid); }

  uint64_t queue_size = g->m_local_in + g->m_local_out;
  q->queue = NULL;
  q->queue_next = NULL;
  q->queue_send = NULL;
  assert(hipMallocManaged(&q->queue, queue_size*sizeof(uint64_t)) == hipSuccess);
  assert(hipMallocManaged(&q->queue_next, queue_size*sizeof(uint64_t)) == hipSuccess);
  assert(hipMallocManaged(&q->queue_send, queue_size*sizeof(uint64_t)) == hipSuccess);

  hipDeviceSynchronize();
  if (q->queue == NULL || q->queue_next == NULL || q->queue_send == NULL)
    throw_err("init_queue_data(), unable to allocate resources\n", procid);

  q->queue_size = 0;
  q->next_size = 0;
  q->send_size = 0;  
  if (debug) { printf("Task %d init_queue_data() success\n", procid); }
}

void clear_queue_data(queue_data_t* q)
{
  if (debug) { printf("Task %d clear_queque_data() start\n", procid); }
  hipFree(q->queue);
  hipFree(q->queue_next);
  hipFree(q->queue_send);
  hipDeviceSynchronize();
  if (debug) { printf("Task %d clear_queque_data() success\n", procid); }
}

void init_comm_data(mpi_data_t* comm)
{
  if (debug) { printf("Task %d init_comm_data() start\n", procid); }

  comm->sendcounts = NULL;
  assert(hipMallocManaged(&comm->sendcounts, nprocs*sizeof(int32_t)) == hipSuccess);
  comm->sendcounts_temp = NULL;
  assert(hipMallocManaged(&comm->sendcounts_temp, nprocs*sizeof(uint64_t)) == hipSuccess);
  comm->recvcounts = NULL;
  assert(hipMallocManaged(&comm->recvcounts, nprocs*sizeof(int32_t)) == hipSuccess);
  comm->recvcounts_temp = NULL;
  assert(hipMallocManaged(&comm->recvcounts_temp, nprocs*sizeof(uint64_t)) == hipSuccess);
  comm->sdispls = NULL;
  assert(hipMallocManaged(&comm->sdispls, nprocs*sizeof(int32_t)) == hipSuccess);
  comm->rdispls = NULL;
  assert(hipMallocManaged(&comm->rdispls, nprocs*sizeof(int32_t)) == hipSuccess);
  comm->rdispls_temp = NULL;
  assert(hipMallocManaged(&comm->rdispls_temp, nprocs*sizeof(uint64_t)) == hipSuccess);
  comm->sdispls_cpy = NULL;
  assert(hipMallocManaged(&comm->sdispls_cpy, nprocs*sizeof(int32_t)) == hipSuccess);
  comm->sdispls_temp = NULL;
  assert(hipMallocManaged(&comm->sdispls_temp, nprocs*sizeof(int64_t)) == hipSuccess);
  comm->sdispls_cpy_temp = NULL;
  assert(hipMallocManaged(&comm->sdispls_cpy_temp, nprocs*sizeof(int64_t)) == hipSuccess);

  hipDeviceSynchronize();
  if (comm->sendcounts == NULL || comm->sendcounts_temp == NULL ||
      comm->recvcounts == NULL || comm->sdispls == NULL || 
      comm->rdispls == NULL || comm->rdispls_temp == NULL ||
      comm->sdispls_cpy == NULL || comm->sdispls_cpy_temp == NULL)
    throw_err("init_comm_data(), unable to allocate resources\n", procid);

  comm->total_recv = 0;
  comm->total_send = 0;
  comm->global_queue_size = 0;
  if (debug) { printf("Task %d init_comm_data() success\n", procid); }
}

void clear_comm_data(mpi_data_t* comm)
{
  if (debug) { printf("Task %d clear_comm_data() start\n", procid); }
  hipFree(comm->sendcounts);
  hipFree(comm->sendcounts_temp);
  hipFree(comm->recvcounts);
  hipFree(comm->recvcounts_temp);
  hipFree(comm->sdispls);
  hipFree(comm->rdispls);
  hipFree(comm->sdispls_cpy);
  hipFree(comm->sdispls_temp);
  hipDeviceSynchronize();
  if (debug) { printf("Task %d clear_comm_data() success\n", procid); }
}

void clear_thread_queue_comm_data(mpi_data_t* comm)
{
  if (debug) { printf("Task %d clear_thread_queue_comm_data() start\n", procid); }
  hipFree(comm->sendcounts);
  hipFree(comm->recvcounts);
  hipFree(comm->sdispls);
  hipFree(comm->rdispls);
  hipFree(comm->sdispls_cpy);
  hipDeviceSynchronize();
  if (debug) { printf("Task %d clear_thread_queue_comm_data() success\n", procid); }
}

void init_thread_queue(thread_queue_t* tq)
{
  tq->tid = omp_get_thread_num();
  if (debug) { 
    printf("Task %d Thread %d init_thread_queue() start\n", procid, tq->tid); 
  }

  tq->thread_queue = (uint64_t*)malloc(THREAD_QUEUE_SIZE*sizeof(uint64_t));
  tq->thread_send = (uint64_t*)malloc(THREAD_QUEUE_SIZE*sizeof(uint64_t));
  if (tq->thread_queue == NULL || tq->thread_send == NULL)
    throw_err("init_thread_queue(), unable to allocate resources\n", procid, tq->tid);

  tq->tid = omp_get_thread_num();
  tq->thread_queue_size = 0;
  tq->thread_send_size = 0;
  if (debug) { 
    printf("Task %d Thread %d init_thread_queue() success\n", procid, tq->tid); 
  }
  
}

void clear_thread_queue(thread_queue_t* tq)
{  
  if (debug) { 
    printf("Task %d Thread %d clear_thread_queue() start\n", procid, tq->tid); 
  }
  free(tq->thread_queue);
  free(tq->thread_send);
  if (debug) { 
    printf("Task %d Thread %d clear_thread_queue() success\n", procid, tq->tid); 
  }
}

void init_thread_comm(thread_comm_t* tc)
{
  tc->tid = omp_get_thread_num();
  if (debug) { 
    printf("Task %d Thread %d init_thread_comm() start\n", procid, tc->tid); 
  }

  tc->v_to_rank = NULL;
  assert(hipMallocManaged(&tc->v_to_rank, nprocs*sizeof(bool))==hipSuccess);
  tc->sendcounts_thread = NULL;
  assert(hipMallocManaged(&tc->sendcounts_thread, nprocs*sizeof(uint64_t))==hipSuccess);
  tc->sendbuf_vert_thread = NULL;
  assert(hipMallocManaged(&tc->sendbuf_vert_thread, THREAD_QUEUE_SIZE*sizeof(uint64_t))==hipSuccess);
  tc->sendbuf_data_thread = NULL;
  assert(hipMallocManaged(&tc->sendbuf_data_thread, THREAD_QUEUE_SIZE*sizeof(uint64_t))==hipSuccess);
  tc->sendbuf_rank_thread = NULL;
  assert(hipMallocManaged(&tc->sendbuf_rank_thread, THREAD_QUEUE_SIZE*sizeof(int32_t))==hipSuccess);
  tc->thread_starts = NULL;
  assert(hipMallocManaged(&tc->thread_starts, nprocs*sizeof(uint64_t))==hipSuccess);
  hipDeviceSynchronize();
  if (tc->v_to_rank == NULL || tc->sendcounts_thread == NULL || 
      tc->sendbuf_vert_thread == NULL || tc->sendbuf_data_thread == NULL || 
      tc->sendbuf_rank_thread == NULL || tc->thread_starts == NULL)
    throw_err("init_thread_comm(), unable to allocate resources\n", procid, tc->tid);

  for (int32_t i = 0; i < nprocs; ++i)
    tc->sendcounts_thread[i] = 0;

  tc->thread_queue_size = 0;

  if (debug) { 
    printf("Task %d Thread %d init_thread_comm() success\n", procid, tc->tid); 
  }
}

void clear_thread_comm(thread_comm_t* tc)
{
  hipFree(tc->v_to_rank);
  hipFree(tc->sendcounts_thread);
  hipFree(tc->sendbuf_vert_thread);
  hipFree(tc->sendbuf_data_thread);
  hipFree(tc->sendbuf_rank_thread);
  hipFree(tc->thread_starts);
  hipDeviceSynchronize();
}

void init_thread_comm_flt(thread_comm_t* tc)
{
  tc->tid = omp_get_thread_num();
  if (debug) { 
    printf("Task %d Thread %d init_thread_comm_flt() start\n", procid, tc->tid); 
  }

  tc->v_to_rank = NULL;
  assert(hipMallocManaged(&tc->v_to_rank, nprocs*sizeof(bool))==hipSuccess);
  tc->sendcounts_thread = NULL;
  assert(hipMallocManaged(&tc->sendcounts_thread, nprocs*sizeof(uint64_t))==hipSuccess);
  tc->sendbuf_vert_thread = NULL;
  assert(hipMallocManaged(&tc->sendbuf_vert_thread, THREAD_QUEUE_SIZE*sizeof(uint64_t))==hipSuccess);
  tc->sendbuf_data_thread_flt = NULL;
  assert(hipMallocManaged(&tc->sendbuf_data_thread_flt, THREAD_QUEUE_SIZE*sizeof(double))==hipSuccess);
  tc->sendbuf_rank_thread = NULL;
  assert(hipMallocManaged(&tc->sendbuf_rank_thread, THREAD_QUEUE_SIZE*sizeof(int32_t))==hipSuccess);
  tc->thread_starts = NULL;
  assert(hipMallocManaged(&tc->thread_starts, nprocs*sizeof(uint64_t))==hipSuccess);
  hipDeviceSynchronize();
  if (tc->v_to_rank == NULL || tc->sendcounts_thread == NULL || 
      tc->sendbuf_vert_thread == NULL || tc->sendbuf_data_thread_flt == NULL || 
      tc->sendbuf_rank_thread == NULL || tc->thread_starts == NULL)
    throw_err("init_thread_comm(), unable to allocate resources\n", procid, tc->tid);

  tc->thread_queue_size = 0;

  for (int32_t i = 0; i < nprocs; ++i)
    tc->sendcounts_thread[i] = 0;

  if (debug) { 
    printf("Task %d Thread %d init_thread_comm_flt() success\n", procid, tc->tid); 
  }
}

void clear_thread_comm_flt(thread_comm_t* tc)
{
  hipFree(tc->v_to_rank);
  hipFree(tc->sendcounts_thread);
  hipFree(tc->sendbuf_vert_thread);
  hipFree(tc->sendbuf_data_thread);
  hipFree(tc->sendbuf_rank_thread);
  hipFree(tc->thread_starts);
  hipDeviceSynchronize();
}


void init_sendbuf_vid_data(mpi_data_t* comm)
{
  if (debug) { printf("Task %d init_sendbuf_vid_data() start\n", procid); }
  
  comm->sdispls_temp[0] = 0;
  comm->sdispls_cpy_temp[0] = 0;
  for (int32_t i = 1; i < nprocs; ++i)
  {
    comm->sdispls_temp[i] = comm->sdispls_temp[i-1] + comm->sendcounts_temp[i-1];
    comm->sdispls_cpy_temp[i] = comm->sdispls_temp[i];
  }

  comm->total_send = comm->sdispls_temp[nprocs-1] + comm->sendcounts_temp[nprocs-1];
  comm->sendbuf_vert = NULL;
  if (comm->total_send != 0){assert(hipMallocManaged(&comm->sendbuf_vert, comm->total_send*sizeof(uint64_t))==hipSuccess);}
  comm->sendbuf_data = NULL;
  if (comm->total_send != 0){assert(hipMallocManaged(&comm->sendbuf_data, comm->total_send*sizeof(uint64_t))==hipSuccess);}
  comm->sendbuf_data_flt = NULL;
  hipDeviceSynchronize(); 
  if ((comm->sendbuf_vert == NULL || comm->sendbuf_data == NULL) && comm->total_send != 0)
    throw_err("init_sendbuf_vid_data(), unable to allocate resources\n", procid);

  comm->global_queue_size = 0;
  uint64_t task_queue_size = comm->total_send;
  MPI_Allreduce(&task_queue_size, &comm->global_queue_size, 1, 
                MPI_UINT64_T, MPI_SUM, MPI_COMM_WORLD);

  if (debug) { printf("Task %d init_sendbuf_vid_data() success\n", procid); }
}

void init_recvbuf_vid_data(mpi_data_t* comm)
{
  if (debug) { printf("Task %d init_recvbuf_vid_data() start\n", procid); }

  for (int32_t i = 0; i < nprocs; ++i)
    comm->recvcounts_temp[i] = 0;

  MPI_Alltoall(comm->sendcounts_temp, 1, MPI_UINT64_T, 
               comm->recvcounts_temp, 1, MPI_UINT64_T, MPI_COMM_WORLD);

  comm->rdispls_temp[0] = 0;

  for (int i = 1; i < nprocs; ++i)
  {
    comm->rdispls_temp[i] = comm->rdispls_temp[i-1] + comm->recvcounts_temp[i-1];
  }

  comm->total_recv = comm->rdispls_temp[nprocs-1] + comm->recvcounts_temp[nprocs-1];
  comm->recvbuf_vert = NULL;
  if (comm->total_recv != 0){assert(hipMallocManaged(&comm->recvbuf_vert, comm->total_recv*sizeof(uint64_t))==hipSuccess);}
  comm->recvbuf_data = NULL;
  if(comm->total_recv != 0){assert(hipMallocManaged(&comm->recvbuf_data, comm->total_recv*sizeof(uint64_t))==hipSuccess);}
  comm->recvbuf_data_flt = NULL;
  hipDeviceSynchronize();
  if ((comm->recvbuf_vert == NULL || comm->recvbuf_data == NULL) && comm->total_recv != 0)
    throw_err("init_recvbuf_vid_data() unable to allocate comm buffers", procid);

  if (debug) { printf("Task %d init_recvbuf_vid_data() success\n", procid); }
}

void init_sendbuf_vid_data_flt(mpi_data_t* comm)
{
  if (debug) { printf("Task %d init_sendbuf_vid_data_flt() start\n", procid); }
  
  comm->sdispls_temp[0] = 0;
  comm->sdispls_cpy_temp[0] = 0;
  for (int32_t i = 1; i < nprocs; ++i)
  {
    comm->sdispls_temp[i] = comm->sdispls_temp[i-1] + comm->sendcounts_temp[i-1];
    comm->sdispls_cpy_temp[i] = comm->sdispls_temp[i];
  }

  comm->total_send = comm->sdispls_temp[nprocs-1] + comm->sendcounts_temp[nprocs-1];
  comm->sendbuf_vert = NULL;
  if (comm->total_send != 0){
    hipError_t err =  hipMallocManaged(&comm->sendbuf_vert, comm->total_send*sizeof(uint64_t));
    if (err != hipSuccess){
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
    assert(err == hipSuccess);
  }
  comm->sendbuf_data = NULL;
  comm->sendbuf_data_flt = NULL;
  if (comm->total_send != 0){assert(hipMallocManaged(&comm->sendbuf_data_flt, comm->total_send*sizeof(double))==hipSuccess);}
  hipDeviceSynchronize();
  if ((comm->sendbuf_vert == NULL || comm->sendbuf_data_flt == NULL) && comm->total_send != 0)
    throw_err("init_sendbuf_vid_data_flt(), unable to allocate resources\n", procid);

  comm->global_queue_size = 0;
  uint64_t task_queue_size = comm->total_send;
  MPI_Allreduce(&task_queue_size, &comm->global_queue_size, 1, 
                MPI_UINT64_T, MPI_SUM, MPI_COMM_WORLD);

  if (debug) { printf("Task %d init_sendbuf_vid_data_flt() success\n", procid); }
}

void init_recvbuf_vid_data_flt(mpi_data_t* comm)
{
  if (debug) { printf("Task %d init_recvbuf_vid_data_flt() start\n", procid); }

  for (int32_t i = 0; i < nprocs; ++i)
    comm->recvcounts_temp[i] = 0;

  MPI_Alltoall(comm->sendcounts_temp, 1, MPI_UINT64_T, 
               comm->recvcounts_temp, 1, MPI_UINT64_T, MPI_COMM_WORLD);

  comm->rdispls_temp[0] = 0;

  for (int i = 1; i < nprocs; ++i)
  {
    comm->rdispls_temp[i] = comm->rdispls_temp[i-1] + comm->recvcounts_temp[i-1];
  }

  comm->total_recv = comm->rdispls_temp[nprocs-1] + comm->recvcounts_temp[nprocs-1];
  comm->recvbuf_vert = NULL;
  if (comm->total_recv != 0){assert(hipMallocManaged(&comm->recvbuf_vert, comm->total_recv*sizeof(uint64_t))==hipSuccess);}
  comm->recvbuf_data = NULL;
  comm->recvbuf_data_flt = NULL;
  if (comm->total_recv != 0){assert(hipMallocManaged(&comm->recvbuf_data_flt, comm->total_recv*sizeof(double))==hipSuccess);}
  hipDeviceSynchronize();
  if ((comm->recvbuf_vert == NULL || comm->recvbuf_data_flt == NULL) && comm->total_recv != 0)
    throw_err("init_recvbuf_vid_data_flt() unable to allocate comm buffers", procid);

  if (debug) { printf("Task %d init_recvbuf_vid_data_flt() success\n", procid); }
}

void clear_recvbuf_vid_data(mpi_data_t* comm)
{
  hipFree(comm->recvbuf_vert);
  hipFree(comm->recvbuf_data);
  hipDeviceSynchronize();

  for (int32_t i = 0; i < nprocs; ++i)
    comm->sendcounts[i] = 0;
  for (int32_t i = 0; i < nprocs; ++i)
    comm->sendcounts_temp[i] = 0;
}

void clear_allbuf_vid_data(mpi_data_t* comm)
{  
  hipFree(comm->sendbuf_vert);
  hipFree(comm->recvbuf_vert);

  if (comm->sendbuf_data != NULL)
    hipFree(comm->sendbuf_data);
  if (comm->recvbuf_data != NULL)
    hipFree(comm->recvbuf_data);
  if (comm->sendbuf_data_flt != NULL)
    hipFree(comm->sendbuf_data_flt);
  if (comm->recvbuf_data_flt != NULL)
    hipFree(comm->recvbuf_data_flt);
hipDeviceSynchronize();

  for (int32_t i = 0; i < nprocs; ++i)
    comm->sendcounts[i] = 0;
  for (int32_t i = 0; i < nprocs; ++i)
    comm->sendcounts_temp[i] = 0;
}
